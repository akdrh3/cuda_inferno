#include "gpu_util.cuh"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
extern "C"
{
#include "util.h"
}
#include <cmath>
#include <climits>
#include <stdio.h>

#include <parallel/algorithm>
#include <omp.h>
#include <algorithm>

int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        printf("Usage: %s <filename> <arraysize in millions>\n", argv[0]);
        return -1;
    }

    const char *file_name = argv[1];
    uint64_t input_size = strtoull(argv[2], NULL, 10) * 1000000;
    int *host_a = (int *)malloc(sizeof(int) * input_size);
    int *host_b = (int *)malloc(sizeof(int) * input_size);
    if (host_a == NULL || host_b == NULL)
    {
        printf("Failed to allocate memory for host array\n");
        return -1;
    }
    read_from_file_cpu(file_name, host_a, input_size);
    int *d_a;
    HANDLE_ERROR(hipMalloc((void **)&d_a, sizeof(int) * input_size));

    uint64_t batch_size = 5 * 1000000000;
    uint64_t numChunks = (input_size + batch_size - 1) / batch_size;
    thrust::device_ptr<int> dev_ptr;

    hipEvent_t event, start, stop, gpu_start, gpu_stop, dtoh_start, dtoh_stop;
    hipEventCreate(&event);

    cuda_timer_start(&start, &stop);
    cuda_timer_start(&gpu_start, &gpu_stop);
    for (uint64_t i = 0; i < numChunks; i++)
    {
        uint64_t left_size = (i < numChunks - 1) ? batch_size : (input_size % batch_size);
        uint64_t offset = i * batch_size;

        HANDLE_ERROR(hipMemcpy(d_a + offset, host_a + offset, left_size * sizeof(int), hipMemcpyHostToDevice));
        dev_ptr = thrust::device_pointer_cast(d_a + offset);
        thrust::sort(dev_ptr, dev_ptr + left_size);
    }
    double gpu_time = cuda_timer_stop(gpu_start, gpu_stop) / 1000.0;
    cuda_timer_start(&dtoh_start, &dtoh_stop);
    HANDLE_ERROR(hipMemcpy(host_b, d_a, input_size * sizeof(int), hipMemcpyDeviceToHost));
    double dtoh_time = cuda_timer_stop(dtoh_start, dtoh_stop) / 1000.0;
    // HANDLE_ERROR(hipMemcpy(host_b, d_a, input_size * sizeof(int), hipMemcpyDeviceToHost));
    print_array_host(host_b, 10);
    printf("sorted : %d \n", isRangeSorted_cpu(host_b, 0, batch_size - 1));

    double total_time = cuda_timer_stop(start, stop) / 1000.0;
    printf("Total time: %lf, gpu sort: %lf, dtoh : %lf\n", total_time, gpu_time, dtoh_time);

    free(host_a);
    free(host_b);
    hipFree(d_a);

    return 0;
}