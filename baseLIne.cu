#include "hip/hip_runtime.h"
#include "gpu_util.cuh"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
extern "C"
{
#include "util.h"
}
#include <cmath>
#include <climits>
#include <stdio.h>

#include <parallel/algorithm>
#include <omp.h>
#include <algorithm>

int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        printf("Usage: %s <filename> <arraysize in millions>\n", argv[0]);
        return -1;
    }

    const char *file_name = argv[1];
    uint64_t input_size = strtoull(argv[2], NULL, 10) * 1000000;
    int *host_a = (int *)malloc(sizeof(int) * input_size);
    int *host_b = (int *)malloc(sizeof(int) * input_size);
    if (host_a == NULL || host_b == NULL)
    {
        printf("Failed to allocate memory for host array\n");
        return -1;
    }
    read_from_file_cpu(file_name, host_a, input_size);
    int *d_a;
    HANDLE_ERROR(hipMalloc((void **)&d_a, sizeof(int) * input_size));

    uint64_t batch_size = 10000000;
    uint64_t numChunks = (input_size + batch_size - 1) / batch_size;
    thrust::device_ptr<int> dev_ptr;

    hipEvent_t event, start, stop, gpu_start, gpu_stop, dtoh_start, dtoh_stop, cpu_start, cpu_stop;
    hipEventCreate(&event);

    cuda_timer_start(&start, &stop);
    cuda_timer_start(&gpu_start, &gpu_stop);
    for (uint64_t i = 0; i < numChunks; i++)
    {
        uint64_t offset = i * batch_size;
        uint64_t left_size = std::min(batch_size, input_size - offset);

        HANDLE_ERROR(hipMemcpy(d_a + offset, host_a + offset, left_size * sizeof(int), hipMemcpyHostToDevice));
        dev_ptr = thrust::device_pointer_cast(d_a + offset);
        thrust::sort(dev_ptr, dev_ptr + left_size);
        HANDLE_ERROR(hipDeviceSynchronize());
    }
    double gpu_time = cuda_timer_stop(gpu_start, gpu_stop) / 1000.0;
    cuda_timer_start(&dtoh_start, &dtoh_stop);
    HANDLE_ERROR(hipMemcpy(host_b, d_a, input_size * sizeof(int), hipMemcpyDeviceToHost));
    double dtoh_time = cuda_timer_stop(dtoh_start, dtoh_stop) / 1000.0;

    cuda_timer_start(&cpu_start, &cpu_stop);
    std::vector<int> merged_result(input_size);
    uint64_t current_offset = 0;

    for (uint64_t i = 0; i < numChunks - 1; i++)
    {
        uint64_t left_size = std::min(batch_size, input_size - current_offset);
        uint64_t next_offset = current_offset + left_size;
        uint64_t next_size = std::min(batch_size, input_size - next_offset);

        std::merge(host_b + current_offset,
                   host_b + current_offset + left_size,
                   host_b + next_offset,
                   host_b + next_offset + next_size,
                   merged_result.begin() + current_offset);
        current_offset += left_size;
    }
    double cpu_time = cuda_timer_stop(cpu_start, cpu_stop) / 1000.0;

    double total_time = cuda_timer_stop(start, stop) / 1000.0;
    printf("Total time: %lf, gpu sort: %lf, dtoh : %lf cpu sort: %lf\n", total_time, gpu_time, dtoh_time, cpu_time);
    printf("sorted : %d \n", isRangeSorted_cpu(host_b, 0, input_size - 1));

    free(host_a);
    free(host_b);
    hipFree(d_a);

    return 0;
}