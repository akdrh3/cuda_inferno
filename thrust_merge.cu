#include "gpu_util.cuh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
extern "C"
{
#include "util.h"
}
#include <cmath>
#include <climits>
#include <stdio.h>

int main(int argc, char *argv[])
{
    // get param from command; filename , arraysize * 1 million
    const char *file_name = argv[1];
    uint64_t input_size = strtoull(argv[2], NULL, 10) * 1000000;
    int *host_a = (int *)malloc(sizeof(int) * input_size);
    if (host_a == NULL)
    {
        return -1;
    }
    read_from_file_cpu(file_name, host_a, input_size);

    size_t pinned_size = 1000000; // Limited by pinned memory size
    size_t numChunks = input_size / pinned_size;

    int *h_aPinned;
    HANDLE_ERROR(hipHostMalloc((void **)&h_aPinned, sizeof(int) * pinned_size));

    int *d_a;
    HANDLE_ERROR(hipMalloc((void **)&d_a, sizeof(int) * input_size));

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    for (size_t i = 0; i < numChunks; i++)
    {
        hipStream_t current_stream = (i % 2 == 0) ? stream1 : stream2;
        int stream_num = (i % 2 == 0) ? 1 : 2;
        printf("current stream : stream%d\n", stream_num);
    }

    free(host_a);
    hipHostFree(h_aPinned);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(d_a);

    return 0;
}
