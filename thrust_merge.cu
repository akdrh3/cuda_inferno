#include "gpu_util.cuh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
extern "C"
{
#include "util.h"
}
#include <cmath>
#include <climits>
#include <stdio.h>

int main(int argc, char *argv[])
{
    if (argc < 3)
    {
        printf("Usage: %s <filename> <arraysize in millions>\n", argv[0]);
        return -1;
    }

    const char *file_name = argv[1];
    uint64_t input_size = strtoull(argv[2], NULL, 10) * 1000000;
    int *host_a = (int *)malloc(sizeof(int) * input_size);
    if (host_a == NULL)
    {
        printf("Failed to allocate memory for host array\n");
        return -1;
    }
    read_from_file_cpu(file_name, host_a, input_size);

    size_t pinned_size = 1000000; // Limited by pinned memory size
    size_t numChunks = (input_size + pinned_size - 1) / pinned_size;

    int *h_aPinned, *h_bPinned;
    HANDLE_ERROR(hipHostMalloc((void **)&h_aPinned, sizeof(int) * pinned_size));
    HANDLE_ERROR(hipHostMalloc((void **)&h_bPinned, sizeof(int) * pinned_size));

    int *d_a;
    HANDLE_ERROR(hipMalloc((void **)&d_a, sizeof(int) * input_size));

    hipStream_t stream1, stream2;
    HANDLE_ERROR(hipStreamCreate(&stream1));
    HANDLE_ERROR(hipStreamCreate(&stream2));

    for (size_t i = 0; i < numChunks; i++)
    {
        size_t current_size = (i < numChunks - 1) ? pinned_size : (input_size % pinned_size);
        size_t bytes = current_size * sizeof(int);
        size_t offset = i * pinned_size;

        int *host_a_src = host_a + offset;
        int *host_a_dst = (i % 2 == 0) ? h_aPinned : h_bPinned;
        int *d_dst = d_a + offset;
        hipStream_t stream_used = (i % 2 == 0) ? stream2 : stream1;

        memcpy(host_a_dst, host_a_src, bytes);
        printf("%d\n", *host_a_dst);

        HANDLE_ERROR(hipMemcpyAsync(d_dst, host_a_dst, bytes, hipMemcpyHostToDevice, stream_used));
    }

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    free(host_a);
    hipHostFree(h_aPinned);
    hipHostFree(h_bPinned);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(d_a);

    return 0;
}
