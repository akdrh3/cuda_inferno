#include "gpu_util.cuh"

__host__ void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__device__ void print_array_device(int *array, int64_t array_size)
{
    for (int64_t i = 0; i < array_size; ++i)
    {
        printf("%d ", array[i]);
    }
    printf("\n");
}

__host__ void cuda_timer_start(hipEvent_t *start, hipEvent_t *stop)
{
    HANDLE_ERROR(hipEventCreate(start));
    HANDLE_ERROR(hipEventCreate(stop));
    HANDLE_ERROR(hipEventRecord(*start, 0));
}

__host__ float cuda_timer_stop(hipEvent_t start, hipEvent_t stop)
{
    float time_elpased;
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time_elpased, start, stop));
    return time_elpased;
}

__device__ int isRangeSorted(int *arr, size_t start, size_t end)
{
    if (start >= end) // Invalid range
    {
        return 1; // A single element or empty range is always sorted
    }

    for (size_t i = start + 1; i < end; ++i)
    {
        if (arr[i - 1] > arr[i])
        {
            return 0; // Found an element out of order
        }
    }
    return 1; // The range is sorted
}