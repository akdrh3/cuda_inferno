#include "gpu_util.cuh"

__host__ void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__device__ void print_array_device(int *array, int64_t array_size){
    for (int64_t i =0; i < array_size; ++i){
        printf("%d ", array[i]);
    }
    printf("\n");
}

__host__ void cuda_timer_start(hipEvent_t *start, hipEvent_t *stop) {
    HANDLE_ERROR(hipEventCreate(start));
    HANDLE_ERROR(hipEventCreate(stop));
    HANDLE_ERROR(hipEventRecord(*start, 0));
}

__host__ float cuda_timer_stop(hipEvent_t start, hipEvent_t stop) {
    float time_elpased;
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&time_elpased, start, stop));
    return time_elpased;
}