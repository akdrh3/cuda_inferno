#include "gpu_util.cuh"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#include "workload.h"
extern "C"
{
#include "util.h"
}
#include <cmath>
#include <climits>
#include <stdio.h>
#include <fstream>
#include <string>

#include <parallel/algorithm>
#include <omp.h>
#include <algorithm>
#include <cstdio>

bool isSorted(const std::vector<double> &data);
void readFileToUnifiedMemory(const char *filename, double *data, uint64_t numElements);
void printSortInfo(struct SortingInfo sortInfo);
void writeToCSV(const std::string &filename, const SortingInfo &SORTINGINFO);

void sortOnCPU(double *start, double *end)
{
    __gnu_parallel::sort(start, end, std::less<double>(), __gnu_parallel::parallel_tag());
}

void sortOnGPU(double *start, double *end)
{
    uint64_t num_items = end - start;
    printf("num_items = %lu\n", num_items);

    // Temporary storage for sorting
    void *d_temp_storage = nullptr;
    uint64_t temp_storage_bytes = 0;

    // Get the amount of temporary storage needed
    hipcub::DeviceRadixSort::SortKeys<double>(d_temp_storage, temp_storage_bytes, start, start, num_items);
    printf("temp_storage_bytes: %lu\n", temp_storage_bytes);

    // Allocate managed memory for temporary storage
    HANDLE_ERROR(hipMallocManaged(&d_temp_storage, temp_storage_bytes));

    // Run the sort operation
    hipcub::DeviceRadixSort::SortKeys<double>(d_temp_storage, temp_storage_bytes, start, start, num_items);

    // Free temporary storage
    hipFree(d_temp_storage);
}

void mergeOnCPU(double *start, double *mid, double *end, double *result)
{
    __gnu_parallel::merge(start, mid, mid, end, result);
}

int main(int argc, char *argv[])
{
    if (argc < 4)
    {
        printf("Usage: %s <filename> <arraysize in millions> <workload on cpu>\n", "./workload");
        return -1;
    }

    const char *file_name = argv[1];
    uint64_t input_size = strtoull(argv[2], NULL, 10) * 1000000;
    float workload_cpu = atof(argv[3]);

    // size_t heapSize = 1L * 1024 * 1024 * 1024;

    // HANDLE_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    double *unSorted = NULL;
    HANDLE_ERROR(hipMallocManaged(&unSorted, input_size * sizeof(double))); // allocate unified memory

    hipEvent_t event, data_trans_start, data_trans_stop, batchSort_start, batchSort_stop, mergeSort_start, mergeSort_stop;
    hipEventCreate(&event);

    cuda_timer_start(&data_trans_start, &data_trans_stop);
    readFileToUnifiedMemory(file_name, unSorted, input_size);

    // Prefetch to GPU before sorting
    HANDLE_ERROR(hipMemPrefetchAsync(unSorted, input_size * sizeof(double), 0, 0));
    double data_trans_time = cuda_timer_stop(data_trans_start, data_trans_stop) / 1000.0;

    uint64_t splitIndex = static_cast<size_t>(workload_cpu * input_size);
    cuda_timer_start(&batchSort_start, &batchSort_stop);
    omp_set_num_threads(16);

    int device_id;
    hipGetDevice(&device_id);
    // Prefetch the GPU part to the GPU
    HANDLE_ERROR(hipMemPrefetchAsync(unSorted + splitIndex, (input_size - splitIndex) * sizeof(double), device_id, 0));

#pragma omp parallel sections
    {
#pragma omp section
        {
            sortOnCPU(unSorted, unSorted + splitIndex);
        }

#pragma omp section
        {
            sortOnGPU(unSorted + splitIndex, unSorted + input_size);
        }
    }
    double batch_sort_time = cuda_timer_stop(batchSort_start, batchSort_stop) / 1000.0;

    // bool sorted;
    // for (uint64_t i = 1; i < input_size; i++)
    // {
    //     if (unSorted[i - 1] > unSorted[i])
    //     {
    //         sorted = false;
    //     }
    //     else sorted = true;
    // }
    // printf("unsorted sorted? : %d \n", sorted);

    cuda_timer_start(&mergeSort_start, &mergeSort_stop);

    // Merging sections (handled on CPU for simplicity)
    double *sortedData = new double[input_size];
    mergeOnCPU(unSorted, unSorted + splitIndex, unSorted + input_size, sortedData);

    double mergeSort_time = cuda_timer_stop(mergeSort_start, mergeSort_stop) / 1000.0;

    SortingInfo SORTINGINFO;
    SORTINGINFO.dataSizeGB = (input_size * sizeof(double)) / (double)(1024 * 1024 * 1024);
    SORTINGINFO.numElements = input_size;
    SORTINGINFO.workload_cpu = workload_cpu;        // Just for reading, adjust according to actual sort
    SORTINGINFO.dataTransferTime = data_trans_time; // Simplified assumption
    SORTINGINFO.batchSortTime = batch_sort_time;
    SORTINGINFO.mergeSortTime = mergeSort_time;
    SORTINGINFO.totalTime = data_trans_time + batch_sort_time + mergeSort_time;
    SORTINGINFO.isSorted = "true"; // isSorted(sortedData); // Update after sorting
    printSortInfo(SORTINGINFO);

    writeToCSV("workload_performance_metrics.csv", SORTINGINFO);
    HANDLE_ERROR(hipFree(unSorted));
    return 0;
}

// Function to read data from the file into unified memory
void readFileToUnifiedMemory(const char *filename, double *data, uint64_t size_of_array)
{
    FILE *file = fopen(filename, "r");
    if (file == NULL)
    {
        fprintf(stderr, "Failed to open file for reading\n");
        exit(EXIT_FAILURE);
    }

    for (uint64_t i = 0; i < size_of_array; i++)
    {
        if (fscanf(file, "%lf", &data[i]) == EOF)
        {
            perror("Error reading from file");
            exit(EXIT_FAILURE);
        }
    }
    fclose(file);
}

void writeToCSV(const std::string &filename, const SortingInfo &SORTINGINFO)
{
    std::ofstream file(filename, std::ios::app);

    std::ifstream testFile(filename);
    bool isEmpty = testFile.peek() == std::ifstream::traits_type::eof();

    // If file is empty, write the header
    if (isEmpty)
    {
        file << "Data Size (GB),Total Elements,CPU workload,Data Transfer Time (s),Batch Sort Time (s),Merge Sort Time (s),Total Time (s), Sorted\n";
    }

    file << SORTINGINFO.dataSizeGB << ","
         << SORTINGINFO.numElements << ","
         << SORTINGINFO.workload_cpu << ","
         << SORTINGINFO.dataTransferTime << ","
         << SORTINGINFO.batchSortTime << ","
         << SORTINGINFO.mergeSortTime << ","
         << SORTINGINFO.totalTime << ","
         << (SORTINGINFO.isSorted ? "Yes" : "No") << "\n";

    file.close();
}

void printSortInfo(struct SortingInfo sortInfo)
{
    printf("Data Size (GB): %.2f\n", sortInfo.dataSizeGB);
    printf("Number of Elements: %zu\n", sortInfo.numElements);
    printf("CPU Workload (%%): %.1f\n", sortInfo.workload_cpu);
    printf("Data Transfer Time (Seconds): %.2f\n", sortInfo.dataTransferTime);
    printf("batch sorting Time (Seconds): %.2f\n", sortInfo.batchSortTime);
    printf("merge sorting Time (Seconds): %.2f\n", sortInfo.mergeSortTime);
    printf("Total Time (Seconds): %.2f\n", sortInfo.totalTime);
    printf("Is Sorted: %s\n", sortInfo.isSorted ? "True" : "False");
}

bool isSorted(const std::vector<double> &data)
{
    for (uint64_t i = 1; i < data.size(); i++)
    {
        if (data[i - 1] > data[i])
        {
            return false;
        }
    }
    return true;
}