#include "hip/hip_runtime.h"

// profiling
int tm();
// data[], size, threads, blocks, 
void mergesort(long*, long, dim3, dim3);
// A[]. B[], size, width, slices, nThreads
__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);

bool verbose = true;
dim3 threadsPerBlock;
dim3 blocksPerGrid;

threadsPerBlock.x = 32;
threadsPerBlock.y = 1;
threadsPerBlock.z = 1;

blocksPerGrid.x = 8;
blocksPerGrid.y = 1;
blocksPerGrid.z = 1;
void mergesort(double* data, uint64_t size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    double* D_data;
    double* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    tm();

    HANDLE_ERROR(hipMallocManaged((void**) &D_data, size * sizeof(double)));
    HANDLE_ERROR(hipMallocManaged((void**) &D_swp, size * sizeof(double)));
    if (verbose)
        std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    //
    // Copy the thread / block info to the GPU as well
    //
    checkCudaErrors(hipMalloc((void**) &D_threads, sizeof(dim3)));
    checkCudaErrors(hipMalloc((void**) &D_blocks, sizeof(dim3)));

    if (verbose)
        std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    checkCudaErrors(hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

    if (verbose)
        std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    double* A = D_data;
    double* B = D_swp;

    uint64_t nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (uint64_t width = 2; width < (size << 1); width <<= 1) {
        uint64_t slices = size / ((nThreads) * width) + 1;

        if (verbose) {
            std::cout << "mergeSort - width: " << width 
                      << ", slices: " << slices 
                      << ", nThreads: " << nThreads << '\n';
            tm();
        }

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        if (verbose)
            std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    // Free the GPU memory
    checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
    if (verbose)
        std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}